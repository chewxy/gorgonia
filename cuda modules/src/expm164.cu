
#include <hip/hip_runtime.h>
#include <math.h>

#ifdef __cplusplus
extern "C" {
#endif


__global__ void expm164(double* A, int size)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int idx = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
	if (idx >= size) {
		return;
	}
	A[idx] = expm1(A[idx]);
}
	
#ifdef __cplusplus
}
#endif