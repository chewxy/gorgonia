
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif


__global__ void sub32(float* A, float* B, int size)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int idx = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
	if (idx >= size) {
		return;
	}
	A[idx] = A[idx] - B[idx]; 
}
	
#ifdef __cplusplus
}
#endif