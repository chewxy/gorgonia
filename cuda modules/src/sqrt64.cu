
#include <hip/hip_runtime.h>
#include <math.h>

#ifdef __cplusplus
extern "C" {
#endif


__global__ void sqrt64(double* A, int size)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int idx = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
	if (idx >= size) {
		return;
	}
	A[idx] = sqrt(A[idx]);
}
	
#ifdef __cplusplus
}
#endif