
#include <hip/hip_runtime.h>
#include <math.h>

#define THREADID \
	int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;\
	int idx = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;

#define CHECKSIZE \
	if (idx >= size) { \
		return; \
	}

#define VVBINOP(name, t, type, op)\
	__global__ void  name ##_vv_ ##t(type* A, type* B, int size) { \
		THREADID \
		CHECKSIZE \
		A[idx] = A[idx] op B[idx];}

#define VSBINOP(name, t, type, op)\
	__global__ void  name ##_vs_ ##t(type* A, type* B, int size) { \
		THREADID \
		CHECKSIZE \
		A[idx] = A[idx] op B[0];}

#define SVBINOP(name, t, type, op)\
	__global__ void  name ##_sv_ ##t(type* A, type* B, int size) { \
		THREADID \
		CHECKSIZE \
		B[idx] = A[0] op B[idx];}

#define SSBINOP(name, t, type, op)\
	__global__ void  name ##_ss_ ##t(type* A, type* B, int size) { \
		THREADID \
		CHECKSIZE \
		A[0] = A[0] op B[0];}

/* VECTOR-VECTOR BIN OP */

extern "C" { VVBINOP(add, f64, double, +) }
extern "C" { VVBINOP(add, f32, float, +) }

extern "C" { VVBINOP(sub, f64, double, -) }
extern "C" { VVBINOP(sub, f32, float, -) }

extern "C" { VVBINOP(mul, f64, double, *) }
extern "C" { VVBINOP(mul, f32, float, *) }

extern "C" { VVBINOP(div, f64, double, /) }
extern "C" { VVBINOP(div, f32, float, /) }

/* VECTOR-SCALAR BIN OP */

extern "C" { VSBINOP(add, f64, double, +) }
extern "C" { VSBINOP(add, f32, float, +) }

extern "C" { VSBINOP(sub, f64, double, -) }
extern "C" { VSBINOP(sub, f32, float, -) }

extern "C" { VSBINOP(mul, f64, double, *) }
extern "C" { VSBINOP(mul, f32, float, *) }

extern "C" { VSBINOP(div, f64, double, /) }
extern "C" { VSBINOP(div, f32, float, /) }

/* SCALAR-VECTOR BIN OP */

extern "C" { SVBINOP(add, f64, double, +) }
extern "C" { SVBINOP(add, f32, float, +) }

extern "C" { SVBINOP(sub, f64, double, -) }
extern "C" { SVBINOP(sub, f32, float, -) }

extern "C" { SVBINOP(mul, f64, double, *) }
extern "C" { SVBINOP(mul, f32, float, *) }

extern "C" { SVBINOP(div, f64, double, /) }
extern "C" { SVBINOP(div, f32, float, /) }

/* SCALAR-SCALAR BIN OP */	

extern "C" { SSBINOP(add, f64, double, +) }
extern "C" { SSBINOP(add, f32, float, +) }

extern "C" { SSBINOP(sub, f64, double, -) }
extern "C" { SSBINOP(sub, f32, float, -) }

extern "C" { SSBINOP(mul, f64, double, *) }
extern "C" { SSBINOP(mul, f32, float, *) }

extern "C" { SSBINOP(div, f64, double, /) }
extern "C" { SSBINOP(div, f32, float, /) }

/* FUNCTION BIN OP */

#define VVFNBINOP(name, t, type, op)\
	__global__ void  name ##_vv_ ##t(type* A, type* B, int size) { \
		THREADID \
		CHECKSIZE \
		A[idx] = op(A[idx], B[idx]);}

#define VSFNBINOP(name, t, type, op)\
	__global__ void  name ##_vs_ ##t(type* A, type* B, int size) { \
		THREADID \
		CHECKSIZE \
		A[idx] = op(A[idx], B[0]);}

#define SVFNBINOP(name, t, type, op)\
	__global__ void  name ##_sv_ ##t(type* A, type* B, int size) { \
		THREADID \
		CHECKSIZE \
		B[idx] = op(A[0], B[idx]);}

#define SSFNBINOP(name, t, type, op)\
	__global__ void  name ##_ss_ ##t(type* A, type* B, int size) { \
		THREADID \
		CHECKSIZE \
		A[0] = op(A[0], B[0]);}

extern "C" { VVFNBINOP(pow, f64, double, pow) }
extern "C" { VVFNBINOP(pow, f32, float, powf) }
extern "C" { VSFNBINOP(pow, f64, double, pow) }
extern "C" { VSFNBINOP(pow, f32, float, powf) }
extern "C" { SVFNBINOP(pow, f64, double, pow) }
extern "C" { SVFNBINOP(pow, f32, float, powf) }
extern "C" { SSFNBINOP(pow, f64, double, pow) }
extern "C" { SSFNBINOP(pow, f32, float, powf) }