
#include <hip/hip_runtime.h>
#include <math.h>

#ifdef __cplusplus
extern "C" {
#endif


__global__ void cos32(float* A, int size)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int idx = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
	if (idx >= size) {
		return;
	}
	A[idx] = cosf(A[idx]);
}
	
#ifdef __cplusplus
}
#endif